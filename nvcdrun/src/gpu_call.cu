#include "hip/hip_runtime.h"
#include "gpu.h"

#include <libnvcd.h>
#include <stdio.h>

LIBNVCD_STORE_FUNCTION_POINTERS_HERE;

extern "C" {  
  __global__ void kernel2() {
    volatile int i = 200000;
    while (i > 0) {
      i--;
    }
  }

  __global__ void kernel3() {
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    if (thread < num_threads) {
      volatile unsigned i = 0;
      while (i < 100000) {
	i++;
      }
    }
  }
  
  __global__ void nvcd_kernel_test() {
    int thread = blockIdx.x * blockDim.x + threadIdx.x;

    int num_threads = blockDim.x * gridDim.x;

    if (thread == 0) {
    
    }

    if (thread < num_threads) {
      volatile int number = 0;

      for (int i = 0; i < 100000; ++i) {
	number += i;
      }
    }
  }
  
  __host__ void gpu_call(unsigned timeflags, unsigned repeat) {

    libnvcd_load();

    puts("=======================================================================");
    printf("[nvcdrun] running test kernels within two separate regions. timeflags = %s\n",
	   libnvcd_time_str(timeflags));
    puts("=======================================================================");
    
    libnvcd_time(timeflags);
    libnvcd_begin("REGION A");    
    
    int num_threads = 1024;
    
    int nblock = 4;
    int threads = num_threads / nblock;

    for (unsigned i = 0; i < repeat; ++i) {
      nvcd_kernel_test<<<nblock, threads>>>();   

      //      num_threads = 2048;

      //      threads = num_threads / nblock;
      //   nvcd_kernel_test<<<nblock, threads>>>();

      kernel3<<<nblock, threads>>>();
    }
    
    libnvcd_end();

    libnvcd_begin("REGION B");
    
    num_threads = 1024;
    
    nblock = 4;
    threads = num_threads / nblock;

    for (unsigned i = 0; i < repeat; ++i) {
      nvcd_kernel_test<<<nblock, threads>>>();   

      num_threads = 2048;

      threads = num_threads / nblock;
      nvcd_kernel_test<<<nblock, threads>>>();

      kernel2<<<nblock, threads>>>();
    }
    
    libnvcd_end();

    puts("[nvcdrun] now for the final kernel run, outside of the test regions");
    
    threads = num_threads / nblock;
    nvcd_kernel_test<<<nblock, threads>>>();

    libnvcd_time_report();
  }
}
