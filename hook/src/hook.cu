#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include <nvcd/commondef.h>
#include <nvcd/util.h>
#include <nvcd/env_var.h>
#include <nvcd/cupti_util.h>
#include <nvcd/nvcd.h>

#include <vector>
#include <unordered_map>
#include <algorithm>
#include <iterator>
#include <memory>
#include <string>
#include <unordered_set>
#include <limits>
#include <type_traits>
#include <sstream>
#include <iomanip>

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <ctype.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/syscall.h>
#include <unistd.h>
#include <errno.h>
#include <dirent.h>
#include <sys/stat.h>
#include <ftw.h>

#include <dlfcn.h>
#include <assert.h>
#include <string.h>
#include <cstdint>

#include <pthread.h>

#include <cupti.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifdef __HIPCC__
#ifndef PRIu64
#define PRIu64 "lu"
#endif

#ifndef PRId64
#define PRId64 "ld"
#endif

#ifndef PRIx64
#define PRIx64 "lx"
#endif

#ifndef PRIu32
#define PRIu32 "u"
#endif

#ifndef PRId32
#define PRId32 "i"
#endif

#ifndef PRIx32
#define PRIx32 "x"
#endif

#endif // __HIPCC__

#define EXTC extern "C"
#define DEV __device__
#define HOST __host__
#define GLOBAL __global__

#define __TONAME___DEV_EXPORT static inline DEV
#define __TONAME___CUDA_EXPORT static inline HOST
#define __TONAME___GLOBAL_EXPORT static GLOBAL

#define STREAM_HEX(bytes) "0x" << std::uppercase << std::setfill('0') << std::setw((bytes) << 1) << std::hex

#define DEV_PRINT_PTR(v) msg_verbosef("&(%s) = %p, %s = %p\n", #v, &v, #v, v)                                                           

using instance_vec_type = std::vector<uint64_t>;
using counter_map_type = std::unordered_map<CUpti_EventID, instance_vec_type>;

instance_vec_type operator - (const instance_vec_type& a, const instance_vec_type& b) {
  ASSERT(a.size() == b.size());
  // in case asserts are disabled (for whatever unlikely reason that may be...)
  size_t sz = std::min(a.size(), b.size());
  instance_vec_type diff(sz, 0);
  for (size_t i = 0; i < sz; ++i) {
    diff[i] = a[i] - b[i];
  }
  return diff;
} 

counter_map_type operator - (const counter_map_type& a, const counter_map_type& b) {
  counter_map_type diff;
  for (const auto& kv: a) {
    const auto& key = kv.first;
    const auto& value = kv.second;
    const instance_vec_type& avec = value;
    // b will be empty the first time this operator overload
    // is called, but this is just a better catch all solution.
    // sure, it's less efficient because b.empty() is simpler to check for.
    // the map is unordered though, so b.find() should have little overhead.
    if (b.find(key) != b.end()) {
      const instance_vec_type& bvec = b.at(key);
      diff[key] = avec - bvec;
    }
    else {
      diff[key] = avec;
    }
  }
  return diff;
}

struct hook_run_info {
  
  counter_map_type counters_start;
  counter_map_type counters_end;
  counter_map_type counters_diff;

  std::string region_name;
  
  size_t curr_num_threads;
  const char* func_name;
  uint32_t run_kernel_exec_count;

  static size_t num_runs;
  
  hook_run_info()
    : curr_num_threads(0),
      func_name(nullptr),
      run_kernel_exec_count(0) {
  }

  ~hook_run_info() {
  }

  void run_kernel_count_inc() {
    run_kernel_exec_count++;
  }
  
  void update(cupti_event_data_t* event_data) {
    ASSERT(curr_num_threads != 0);   

    curr_num_threads = 0;
    run_kernel_exec_count = 0;
   

    // we do this to compute the difference
    // from the previous run
    for (const auto& kv: counters_end) {
      counters_start[kv.first] = kv.second;      
    }
    
    cupti_event_data_enum_event_counters(event_data,
                                         this,
                                         hook_run_info::enum_event_counters);    
    
    counters_diff = counters_end - counters_start;

    num_runs++;
  }

  static bool enum_event_counters(cupti_enum_event_counter_iteration_t* it) {
    hook_run_info* run_info = static_cast<hook_run_info*>(it->user_param);
    if (run_info->counters_end[it->event].empty()) {
      run_info->counters_end[it->event].resize(it->num_instances, 0);
    }
    ASSERT(it->instance < it->num_instances);
    run_info->counters_end[it->event][it->instance] += it->value;
    return true;
  }
  
  void report(cupti_event_data_t* event_data) {
    ASSERT(num_runs > 0);
    
    msg_userf("================================ invocation %" PRIu64 " for \'%s\' ================================\n",
	      num_runs - 1,
	      region_name.c_str());

    std::stringstream ss;
    msg_verbosef("counters_diff size: %" PRIu64 "\n", counters_diff.size());
    for (const auto& kv : counters_diff) {
      const auto& key = kv.first;
      const auto& value = kv.second;
      ASSERT(!value.empty());
      char* event_name = cupti_event_get_name(key);
      ASSERT(event_name != nullptr);
      double avg = 0;
      uint64_t summation = 0;
      uint64_t maximum = 0; // the lowest possible count
      uint64_t minimum = std::numeric_limits<uint64_t>::max(); //something very large so that it changes
      uint64_t temp_var = 0;
      for (size_t index = 0; index < value.size(); ++index) {
	temp_var = value.at(index);
	summation += temp_var;
	avg += temp_var;
	maximum = (maximum < temp_var) ? temp_var : maximum;
	minimum = (minimum > temp_var) ? temp_var : minimum;
      }
      avg /= static_cast<double>(value.size());
      ss << "|COUNTER|" << region_name << ":" << event_name << ": SUM: " << summation << " AVG: " << avg << " MAX: " << maximum << " MIN: " << minimum << "\n";
      free(event_name);
    }
    
    msg_userf("%s", ss.str().c_str());
    
    cupti_report_event_data(event_data);
  }
};

size_t hook_run_info::num_runs = 0;

static hook_run_info* g_run_info = nullptr;

__TONAME___CUDA_EXPORT void __toname___report() {
  ASSERT(g_run_info != nullptr);    
  g_run_info->report(nvcd_get_events());
}

__TONAME___CUDA_EXPORT void __toname___init() {
  nvcd_init_cuda();

  if (g_run_info == nullptr) {
    g_run_info = new hook_run_info();
  }
        
  ASSERT(g_nvcd.initialized == true);
  ASSERT(g_run_info != nullptr);
}

__TONAME___CUDA_EXPORT void __toname___host_begin(const char* region_name, int num_cuda_threads) {     
  __toname___init();

  g_run_info->region_name = std::string(region_name);

  ASSERT(g_nvcd.initialized == true);
  ASSERT(g_run_info != nullptr);

  g_run_info->curr_num_threads = static_cast<size_t>(num_cuda_threads);

  nvcd_init_events(g_nvcd.devices[0],
                   g_nvcd.contexts[0]);
}

__TONAME___CUDA_EXPORT bool __toname___host_finished() {
  return cupti_event_data_callback_finished(nvcd_get_events());
}

__TONAME___CUDA_EXPORT void __toname___terminate();

__TONAME___CUDA_EXPORT void __toname___host_end() {
  ASSERT(g_nvcd.initialized == true);
    
  nvcd_calc_metrics();

  g_run_info->update(nvcd_get_events());

  __toname___report();   

  __toname___terminate();
}
 

__TONAME___CUDA_EXPORT void __toname___terminate() {
  nvcd_reset_event_data();
 
  for (int i = 0; i < g_nvcd.num_devices; ++i) {
    ASSERT(g_nvcd.contexts[i] != nullptr);
    safe_free_v(g_nvcd.device_names[i]);
            
    if (g_nvcd.contexts_ext[i] == false) {
      CUDA_DRIVER_FN(hipCtxDestroy(g_nvcd.contexts[i]));
    }
  }

  safe_free_v(g_nvcd.device_names);
  safe_free_v(g_nvcd.devices);
  safe_free_v(g_nvcd.contexts);

  g_nvcd.initialized = false;
}

#define NVCD_TIMEFLAGS_NONE 0
#define NVCD_TIMEFLAGS_REGION (1 << 2)
#define NVCD_TIMEFLAGS_KERNEL (1 << 1)
#define NVCD_TIMEFLAGS_RUN (1 << 0)

enum spec
  {
   f_000 = NVCD_TIMEFLAGS_NONE,
   f_00r = NVCD_TIMEFLAGS_RUN,
   f_0k0 = NVCD_TIMEFLAGS_KERNEL,
   f_0kr = NVCD_TIMEFLAGS_KERNEL | NVCD_TIMEFLAGS_RUN,
   f_r00 = NVCD_TIMEFLAGS_REGION,
   f_r0r = NVCD_TIMEFLAGS_REGION | NVCD_TIMEFLAGS_RUN,
   f_rk0 = NVCD_TIMEFLAGS_REGION | NVCD_TIMEFLAGS_KERNEL,
   f_rkr = NVCD_TIMEFLAGS_REGION | NVCD_TIMEFLAGS_KERNEL | NVCD_TIMEFLAGS_RUN       
  };

struct timeflags { 
private:
  uint32_t value;
  
public:
  timeflags(uint32_t value)    
    : value(check(value, f_000, f_rkr))
  {}

  uint32_t check(uint32_t x, uint32_t min, uint32_t max) {
    if (!(min <= x && x <= max)) {
      exit_msg(stdout,
	       EBAD_INPUT,
	       "[HOOKE ERROR] value = %" PRIu32 " is out of defined range [%" PRIu32 ", %" PRIu32 "].\n",
	       x,
	       min,
	       max);
    }
    return x;
  }

  operator uint32_t () const {
    return value;
  }
};


typedef std::unordered_map<uint32_t, std::vector<std::string>> time_output_map_type;

static time_output_map_type time_map
  {
   {
    f_00r,
    {
     "run"
    }
   },

   {
    f_0k0,
    {
     "kernel"
    }
   },

   {
    f_0kr,
    {
     "kernel",
     "run"
    }
   },

   {
    f_r00,
    {
     "region"
    }
   },

   {
    f_r0r,
    {
     "region",
     "run"
    }
   },

   {
    f_rk0,
    {
     "region",
     "kernel"
    }
   },

   {
    f_rkr,
    {
     "region",
     "kernel",
     "run"
    }
   }
  };

static bool g_enabled = false;

struct timeslice {
  struct timespec start;
  struct timespec end;
  double time;
  bool set;

  timeslice() { reset(); }

  timeslice& reset() {
    start.tv_sec = start.tv_nsec = 0;
    end.tv_sec = end.tv_nsec = 0;
    time = 0.0;
    set = false;
    return *this;
  }

  timeslice& go() {
    clock_gettime(CLOCK_REALTIME, &start);
    return *this;
  }

  double seconds(struct timespec* t) const { return (double)t->tv_sec + ((double)t->tv_nsec) * 1e-9; }

  timeslice& stop() {
    if (!set) {
      clock_gettime(CLOCK_REALTIME, &end);
      time = seconds(&end) - seconds(&start);
    }
    return *this;
  }

  operator double () const {
    return time;
  }
};


struct timetree {
  enum ttype
    {
     node,
     leaf
    };

  const ttype type;
  
  timetree(ttype type) : type(type) {}
  
  using ptr_type = std::unique_ptr<timetree>;
  
  virtual double value() const = 0;

  virtual std::string to_string(const std::string& region, timeflags flags, uint32_t depth) const {
    std::stringstream ret;
    std::string title{time_map.at(flags).at(depth)};
    std::string tabs = (depth != 0) ? std::string(depth, '\t') : "";
    ret << tabs << "[HOOK TIME " << ((title == "region") ? ("region " + region) : title) <<  "] " << value() << " seconds\n";
    return ret.str();
  }
};

struct timenode : public timetree {
  std::vector<timetree::ptr_type> children;

  timenode() : timetree(node) {}
  
  double value() const override {
    double ret = 0.0;
    for (const auto& child: children) {
      ret += child->value();
    }    
    return ret;
  }

  std::string to_string(const std::string& region, timeflags flags, uint32_t depth) const override {
    std::stringstream ret;
    ret << timetree::to_string(region, flags, depth);
    for (const auto& child: children) {
      ret << child->to_string(region, flags, depth + 1);
    }
    return ret.str();
  }
};

struct timeleaf : public timetree {
  timeslice v;

  timeleaf() : timetree(leaf) {}

  double value() const override {
    return static_cast<double>(v);
  }
};

struct hook_time_record {
  std::string region_name;
  std::vector<std::string> dumps;
};

static std::vector<hook_time_record> g_time_records;

using call_interval_type = int32_t;

static constexpr call_interval_type k_max_call_interval{10000};
static constexpr call_interval_type k_min_call_interval{0};
static constexpr call_interval_type k_unset_call_interval{-1};

struct kernel_interval_params {
  static call_interval_type interval;
  call_interval_type call_count;
  
  kernel_interval_params()
    : call_count(0) {

    ASSERT((interval == k_unset_call_interval) ||
	   (k_min_call_interval <= interval &&
	    interval <= k_max_call_interval));
    
    if (interval == k_unset_call_interval) {
      char* interval_str = getenv(ENV_SAMPLE);

      if (interval_str != nullptr) {
	bool ok = false;
	// we restrict ourselves currently to a single value
	char* end_ptr = nullptr;
	call_interval_type ci = strtol(interval_str, &end_ptr, 10);
	  
	ok =
	  C_ASSERT(k_min_call_interval <= ci) &&
	  C_ASSERT(ci <= k_max_call_interval) &&
	  // ensures the entire string is a valid base 10 integer
	  C_ASSERT(end_ptr[0] == '\0' &&
		   interval_str[0] != '\0');
	  
	if (ok) {
	  interval = ci;
	}
	else {
	  interval = k_min_call_interval;
	}
      }
      else {
	interval = k_min_call_interval;
      }

      printf("[HOOK CALL INTERVAL = %" PRId32"]\n", interval);
    }
  }
};

call_interval_type kernel_interval_params::interval{k_unset_call_interval};

static std::unordered_map<uintptr_t, kernel_interval_params>  g_call_counts;

namespace {
  struct push {
    timetree::ptr_type& src;

    void to(timetree::ptr_type& dst) {
      ASSERT(dst->type == timetree::node);
      dynamic_cast<timenode*>(dst.get())->children.push_back(std::move(src));
    }
  };
  
  struct stop {
    timetree::ptr_type& src;

    stop(timetree::ptr_type& src)
      : src(src)
    {
      ASSERT(src->type == timetree::leaf);
      dynamic_cast<timeleaf*>(src.get())->v.stop();
    }
    
    void then_push_to(timetree::ptr_type& dst) {
      push { src }.to(dst);
    }
  };

  template <class T>
  struct set_traits {
    timetree::ptr_type& v;
    set_traits(timetree::ptr_type& v) : v(v) { v.reset(new T()); }   
  };

  template <class T>
  struct set_to : public set_traits<T> {
    set_to(timetree::ptr_type& v) : set_traits<T>(v) {}
    
    void then_start() {
      exit_msg(stdout,
	       EBAD_PATH,
	       "%s\n",
	       "[HOOK ERROR] then_start() reached with invalid type.\n");      
    }
  };

  template <>
  struct set_to<timeleaf> : public set_traits<timeleaf> {
    set_to(timetree::ptr_type& v) : set_traits<timeleaf>(v) {}
    
    void then_start() {
      dynamic_cast<timeleaf*>(v.get())->v.go();
    }
  };

  const std::vector<timetree::ptr_type>& get_children(const timetree::ptr_type& parent) {
    return dynamic_cast<timenode*>(parent.get())->children;
  }

  struct add {
    const std::string& dump;

    void to(const std::string& region_name) {
      bool found = false;
      for (auto& entry: g_time_records) {
        found = entry.region_name == region_name;
        if (found) {
          entry.dumps.push_back(dump);
          break;
        }
      }
      if (!found) {
        g_time_records.push_back(hook_time_record {region_name, {dump}});
      }
    }
  };

  struct call_for {
    uintptr_t symaddr;

    call_for(const void* func)
      : symaddr{reinterpret_cast<uintptr_t>(func)}{    
    }
  
    bool is_ready() {
      bool ready = (g_call_counts[symaddr].call_count % kernel_interval_params::interval) == 0;
      g_call_counts[symaddr].call_count++;
      return ready;
    }    
  };
}

struct hook_time_info {
  timetree::ptr_type root;
  
  timetree::ptr_type region;
  timetree::ptr_type kernel;
  timetree::ptr_type run;

  std::string region_name;
  
  timeflags flags;

  hook_time_info ()
    : root(nullptr),
      region(nullptr),
      kernel(nullptr),
      run(nullptr),
      flags(0)
  {
    set_to<timenode>{root};
  }

  uint32_t f32() const { return static_cast<uint32_t>(flags); }
  
  bool test(uint32_t f) const { return (f32() & f) == f; }
  
  bool has_region() const { return test(NVCD_TIMEFLAGS_REGION); }
  bool has_kernel() const { return test(NVCD_TIMEFLAGS_KERNEL); }
  bool has_run() const { return test(NVCD_TIMEFLAGS_RUN); }
  
  void begin_region(const char* region_name) {
    this->region_name = std::string(region_name);
    if (has_region()) {
      switch (f32()) {
      case f_rkr:
      case f_rk0:
      case f_r0r:
	set_to<timenode>{region};
	break;
      case f_r00:
	set_to<timeleaf>{region}.then_start();
	break;
      }
    }
  }

  void begin_kernel() {
    if (has_kernel()) {
      switch (f32()) {
      case f_rkr:
      case f_0kr:
	set_to<timenode>{kernel};
	break;
	
      case f_rk0:
      case f_0k0:
	set_to<timeleaf>{kernel}.then_start();
	break;
      }
    }    
  }
  
  void begin_run() {
    if (has_run()) {
      set_to<timeleaf>{run}.then_start();
    }
  }

  void end_run() {
    if (has_run()) {

      switch (f32()) {
      case f_rkr:
      case f_0kr:
	stop{run}.then_push_to(kernel);
	break;
      case f_r0r:
	stop{run}.then_push_to(region);
	break;
      case f_00r:
	stop{run}.then_push_to(root);
	break;
      }
    }
  }

  void end_kernel() {
    if (has_kernel()) {
      switch (f32()) {
      case f_rk0:
	stop{kernel}.then_push_to(region);	
	break;

      case f_0k0:
	stop{kernel}.then_push_to(root);
	break;
	
      case f_0kr:
	push{kernel}.to(root);
	break;
	
      case f_rkr:
	push{kernel}.to(region);
	break;
      }
    }
  }

  void end_region() {
    if (has_region()) {
      switch (f32()) {
      case f_r00:
	stop{region}.then_push_to(root);
	
	break;
      case f_r0r:
      case f_rk0:
      case f_rkr:
	push{region}.to(root);
	break;
      }
    }
  }

  void record() {
    ASSERT(!region_name.empty());
    std::stringstream ss;
    for (const auto& child_ptr: get_children(root)) {
      ss << child_ptr->to_string(region_name, flags, 0);
    }
    add{ss.str()}.to(region_name);
  }
};

static std::unique_ptr<hook_time_info> g_timer{nullptr};

static void reset_timer() {
  ASSERT(!g_enabled);
  if (!g_enabled) {
    if (g_timer) {
      timeflags tmp(g_timer->flags);
      g_timer.reset(new hook_time_info());
      g_timer->flags = tmp;
    }
  }
}

template <class TKernFunType,
	  class ...TArgs>
static inline hipError_t nvcd_run_metrics2(const TKernFunType& kernel, 				     
					    TArgs... args) {
  cupti_event_data_t* __e = nvcd_get_events();                           
  
  ASSERT(__e->is_root == true);                                       
  ASSERT(__e->initialized == true);                                   
  ASSERT(__e->metric_data != nullptr);                                   
  ASSERT(__e->metric_data->initialized == true);                      

  hipError_t result = hipSuccess;
  for (uint32_t i = 0; result == hipSuccess && i < __e->metric_data->num_metrics; ++i) {      
    cupti_event_data_begin(&__e->metric_data->event_data[i]);         

    while (result == hipSuccess && !cupti_event_data_callback_finished(&__e->metric_data->event_data[i])) {
      if (g_timer) g_timer->begin_run();
      kernel(args...);                       
      CUDA_RUNTIME_FN(hipDeviceSynchronize());
      if (g_timer) g_timer->end_run();
      g_run_info->run_kernel_count_inc();				
    }                                                                 
                                                                        
    cupti_event_data_end(&__e->metric_data->event_data[i]);
  }

  return result;
}


template <class TKernFunType, class ...TArgs>
static inline hipError_t nvcd_run2(const TKernFunType& kernel, 
				    TArgs... args) {

  hipError_t result = hipSuccess;

  if (nvcd_has_events()) {
    cupti_event_data_begin(nvcd_get_events());  
    while (result == hipSuccess && !__toname___host_finished()) {
      if (g_timer) g_timer->begin_run();
      result = kernel(args...);                       
      CUDA_RUNTIME_FN(hipDeviceSynchronize());
      if (g_timer) g_timer->end_run();
      g_run_info->run_kernel_count_inc();			
    }                                                                   
    cupti_event_data_end(nvcd_get_events());
  }

  if (result == hipSuccess && nvcd_has_metrics()) {  
    result = nvcd_run_metrics2(kernel, args...);
  }

  return result;
}

typedef hipError_t (*cudaLaunchKernel_fn_t)(const void* func, dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream);

class hook_driver {
public:
  using host_thread_id_type = pthread_t;

  hook_driver(host_thread_id_type htid)
    : m_run_info(new hook_run_info()),
      m_host_thread_id(htid),
      m_hook_enabled(false)
  {}
  
  void event_trace_begin(int num_cuda_threads, hipStream_t stream) {
    ASSERT(!m_region_name.empty());
    ASSERT(num_cuda_threads > 0);
    
    m_run_info->region_name = m_region_name;
    m_run_info->curr_num_threads = static_cast<size_t>(num_cuda_threads);
    
    m_per_trace.begin(stream);
  }

  template <class ...TArgs>
  hipError_t run(cudaLaunchKernel_fn_t real_cudaLaunchKernel, TArgs... args) {
    hipError_t result = hipSuccess;

    auto run_trace_loop_for = [&](cupti_event_data_t* e) {
      cupti_event_data_begin(e);
      while (result == hipSuccess && !cupti_event_data_callback_finished(e)) {
        result = real_cudaLaunchKernel(args...);
        CUDA_RUNTIME_FN(hipDeviceSynchronize());
        m_run_info->run_kernel_count_inc();
      }
      cupti_event_data_end(e);
    };
    
    if (event_data()->has_events) {
      run_trace_loop_for(event_data());
    }

    if (result == hipSuccess && event_data()->has_metrics) {  
      ASSERT(event_data()->is_root == true);                                       
      ASSERT(event_data()->initialized == true);                                   
      ASSERT(event_data()->metric_data != nullptr);                                   
      ASSERT(event_data()->metric_data->initialized == true);                      

      cupti_event_data_t* metric_event_buffer =
        & (event_data()->metric_data->event_data[0]);

      for (uint32_t i = 0;
           result == hipSuccess &&
             i < event_data()->metric_data->num_metrics;
           ++i) {
        run_trace_loop_for(&metric_event_buffer[i]);
      }
    }

    return result;
  }
  
  void event_trace_end() {
    m_per_trace.calc_metrics();
    
    m_run_info->update(event_data());
    m_run_info->report(event_data());
    
    m_per_trace.end();
  }
  
  void region_begin(std::string region_name) {
    m_region_name = std::move(region_name);
  }

  void region_end() {
    
  }

  
private:
    struct per_trace {
      cupti_event_data_t m_cupti_event_data;
      hipCtx_t m_cu_context;
      hipDevice_t m_cu_device;
      hipStream_t m_cuda_stream;
      int m_device;
    
      bool m_cu_context_is_creat;
    
      per_trace() {
        reset();
      }

      void reset() {
        cupti_event_data_set_null(&m_cupti_event_data);
        m_cu_context = nullptr;
        m_cu_device = -1;
        m_cuda_stream = nullptr;
        m_device = -1;
        m_cu_context_is_creat = false;
      }

      void begin(hipStream_t stream) {
        m_cuda_stream = stream;
      
        CUDA_RUNTIME_FN(hipGetDevice(&m_device));
    
        CUDA_DRIVER_FN(hipDeviceGet(&m_cu_device, m_device));

        CUDA_DRIVER_FN(hipCtxGetCurrent(&m_cu_context));
        
        if (m_cu_context == nullptr) {
          CUDA_DRIVER_FN(hipCtxCreate(&m_cu_context,
                                     0,
                                     m_cu_device));

          m_cu_context_is_creat = true;
        }

        m_cupti_event_data.cuda_context = m_cu_context;
        m_cupti_event_data.cuda_device = m_cu_device;
        m_cupti_event_data.is_root = true;

        cupti_event_data_init(&m_cupti_event_data);
      }

      void calc_metrics() {
        if (m_cupti_event_data.has_metrics) {
          cupti_event_data_calc_metrics(&m_cupti_event_data);
        }
      }

      void end() {
        cupti_event_data_free(&m_cupti_event_data);
        if (m_cu_context_is_creat) {
          ASSERT(m_cu_context != nullptr);
          CUDA_DRIVER_FN(hipCtxDestroy(m_cu_context));
        }
        reset();
      }
    };
  
  cupti_event_data_t* event_data() { return &m_per_trace.m_cupti_event_data; }
  
  per_trace m_per_trace;
  
  std::unique_ptr<hook_run_info> m_run_info;

  std::string m_region_name;
  
  host_thread_id_type m_host_thread_id;
  
  bool m_hook_enabled;
};

class hook_driver_manager {
public:
  hook_driver* driver_for(hook_driver::host_thread_id_type thread) {
    if (!m_hooks[thread]) {
      m_hooks[thread].reset(new hook_driver(thread));
    }
    return m_hooks.at(thread).get();
  }
  
private:
  std::unordered_map<hook_driver::host_thread_id_type,
                     std::unique_ptr<hook_driver>> m_hooks;
};

static std::unique_ptr<hook_driver_manager> g_drvman(new hook_driver_manager());

C_LINKAGE_START

static char g_region_buffer[256] = {0};

static cudaLaunchKernel_fn_t real_cudaLaunchKernel = nullptr;

void print_func(const void* func) {
  const char* f = static_cast<const char*>(func);
  printf("[HOOK INFO - func: string = %s, address %p" PRIx64 "]\n", f, func);
}

NVCD_EXPORT __host__ hipError_t hipLaunchKernel(reinterpret_cast<const void*>(const void* func),
						  dim3 gridDim,
						  dim3 blockDim,
						  void** args,
						  size_t sharedMem,
						  hipStream_t stream) {
  hipError_t ret = hipSuccess;
  
  if (real_cudaLaunchKernel == nullptr) {
    real_cudaLaunchKernel = (cudaLaunchKernel_fn_t) dlsym(RTLD_NEXT, "hipLaunchKernel");
  }
  if (g_enabled) {
    if (call_for(func).is_ready()) {
      printf("[HOOK ON %s - %s; symbol = %p]\n", __FUNC__, g_region_buffer, func);
      if (g_timer) {
	g_timer->begin_kernel();
      }
      __toname___host_begin(g_region_buffer, gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z);
      ret = nvcd_run2(real_cudaLaunchKernel, func, gridDim, blockDim, args, sharedMem, stream);
      __toname___host_end();
      if (g_timer) {
	g_timer->end_kernel();
      }
    }
  }
  else {
    printf("[HOOK OFF %s]\n", __FUNC__);
    ret = real_cudaLaunchKernel(func, gridDim, blockDim, args, sharedMem, stream);
  }
  //  print_func(func);
  return ret;
}

NVCD_EXPORT void libnvcd_time(uint32_t flags) {
  // We absolutely don't want to mess with the timer state
  // if a region has been enabled.
  ASSERT(!g_enabled);
  if (!g_enabled) {
    // user disabled timer, so we'll refrain from
    // continuing to record.
    if (flags == 0) {
      g_timer.reset(nullptr);
    } else {
      g_timer.reset(new hook_time_info());
      g_timer->flags = timeflags(flags);
    }
  }
}

NVCD_EXPORT void libnvcd_time_report() {
  std::stringstream ss;
  for (const auto& region_entries: g_time_records) {
    for (const auto& dump: region_entries.dumps) {
      ss << dump;
    }
  }
  printf("%s\n", ss.str().c_str());
  g_time_records.clear();
}

NVCD_EXPORT void libnvcd_begin(const char* region_name) {
  // a null region name is totally useless,
  // and will also likely create a segfault,
  // so we may as well enforce non-null input.
  ASSERT(region_name != nullptr);
  ASSERT(strlen(region_name) <= 256);
  if (region_name != nullptr) {
    strncpy(g_region_buffer, region_name, 255);
    g_enabled = true;
    if (g_timer) {
      g_timer->begin_region(region_name);
    }
  }
}

NVCD_EXPORT void libnvcd_end() {
  // g_enabled == false implies a significant flaw
  // in the program logic of the caller.
  // It also opens the door to further errors that
  // coulud arise internally in the future.
  ASSERT(g_enabled == true);
  if (g_enabled) {
    if (g_timer) { 
      g_timer->end_region();
      g_timer->record();   
    }
    // make sure this is set to false before
    // reset_timer() is called
    g_enabled = false;
    reset_timer();
    hook_run_info::num_runs = 0;
  }
}

C_LINKAGE_END
